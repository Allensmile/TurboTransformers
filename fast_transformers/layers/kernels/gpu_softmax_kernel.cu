#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <numeric>
#include "fast_transformers/layers/kernels/gpu_common.h"
#include "fast_transformers/layers/kernels/gpu_softmax_kernel.h"

namespace fast_transformers {
namespace layers {
namespace kernels {

// unroll for loop using unroll size as 4.
// blk_size can be arbitary positive intergers.
// For sake of the effecieny blk_size should better be 4x.
__global__ void softmax_kernel_unroll4(float* qk_buf_, const float* attr_mask,
                                       int batch_size, int head_num,
                                       int seq_len, float scaler,
                                       int blk_size) {
  int batch_id = blockIdx.x * blk_size / seq_len / head_num;
  int tid = threadIdx.x;
  int qk_offset = tid + blockIdx.x * seq_len * blk_size;

  const int loop_unroll_size = 4;
  static __shared__ float s_sum[loop_unroll_size];
  float qk_list[loop_unroll_size];
  float qk_sum_list[loop_unroll_size];
  const int max_blk_inner_size = loop_unroll_size;

  float mask_val = tid < seq_len ? attr_mask[tid + batch_id * seq_len] : 0.0f;
  float qk;
  int i;
  for (i = 0; i < blk_size / max_blk_inner_size * max_blk_inner_size;
       i += max_blk_inner_size) {
    int blk_size_inner = max_blk_inner_size;
    if (threadIdx.x < seq_len) {
      int qk_buf_offset = qk_offset + i * seq_len;
      qk = qk_buf_[qk_buf_offset];
      qk_sum_list[0] = qk_list[0] = __expf((qk * scaler + mask_val));

      qk = qk_buf_[qk_buf_offset + seq_len];
      qk_sum_list[1] = qk_list[1] = __expf((qk * scaler + mask_val));

      qk = qk_buf_[qk_buf_offset + 2 * seq_len];
      qk_sum_list[2] = qk_list[2] = __expf((qk * scaler + mask_val));

      qk = qk_buf_[qk_buf_offset + 3 * seq_len];
      qk_sum_list[3] = qk_list[3] = __expf((qk * scaler + mask_val));
    } else {
      qk_sum_list[0] = qk_list[0] = 0.0;
      qk_sum_list[1] = qk_list[1] = 0.0;
      qk_sum_list[2] = qk_list[2] = 0.0;
      qk_sum_list[3] = qk_list[3] = 0.0;
    }
    blockReduceSum_Elem4(qk_sum_list);
    if (tid == 0) {
      s_sum[0] = qk_sum_list[0] + 1e-6f;
      s_sum[1] = qk_sum_list[1] + 1e-6f;
      s_sum[2] = qk_sum_list[2] + 1e-6f;
      s_sum[3] = qk_sum_list[3] + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len) {
      qk_buf_[qk_offset + seq_len * (i + 0)] = (qk_list[0] / s_sum[0]);
      qk_buf_[qk_offset + seq_len * (i + 1)] = (qk_list[1] / s_sum[1]);
      qk_buf_[qk_offset + seq_len * (i + 2)] = (qk_list[2] / s_sum[2]);
      qk_buf_[qk_offset + seq_len * (i + 3)] = (qk_list[3] / s_sum[3]);
    }  // endif
  }    // for i

  // dealing with the reminding lines
  int blk_size_inner = blk_size % max_blk_inner_size;
  if (blk_size_inner == 0) return;
  if (threadIdx.x < seq_len) {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk = qk_buf_[qk_offset + (i + j) * seq_len];
      qk_sum_list[j] = qk_list[j] = __expf((qk * scaler + mask_val));
    }
  } else {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk_sum_list[j] = qk_list[j] = 0.0;
    }
  }
  for (int j = 0; j < blk_size_inner; ++j) {
    qk_sum_list[j] = blockReduceSum(qk_list[j]);
    __syncthreads();
  }
  if (tid == 0) {
    for (int j = 0; j < blk_size_inner; ++j) {
      s_sum[j] = qk_sum_list[j] + 1e-6f;
    }
  }
  __syncthreads();
  if (threadIdx.x < seq_len) {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk_buf_[qk_offset + seq_len * (i + j)] = (qk_list[j] / s_sum[j]);
    }
  }  // endif
}

// nvidia version block size on the high dimension is always 1, may lead to
// low occupancy
__global__ void softmax_kernel_noblk(float* qk_buf_, const float* attr_mask,
                                     const int batch_size, const int head_num,
                                     const int seq_len, const float scaler) {
  int batch_id = blockIdx.x / head_num / seq_len;
  int qk_offset = blockIdx.x * seq_len;
  int mask_offset = batch_id * seq_len;

  __shared__ float s_sum, s_max;

  float qk = threadIdx.x < seq_len ? qk_buf_[threadIdx.x + qk_offset] : 0.0f;
  float mask_val =
      threadIdx.x < seq_len ? attr_mask[threadIdx.x + mask_offset] : 0.0f;

  // mask_val = (1.0f - mask_val) * -10000.0f;
  float tmp = threadIdx.x < seq_len ? (qk * scaler + mask_val) : -1e20f;
  float max_val = blockReduceMax(tmp);
  if (threadIdx.x == 0) s_max = max_val;
  __syncthreads();

  float qk_tmp = threadIdx.x < seq_len ? __expf((tmp - s_max)) : 0.0f;
  float sum_val = blockReduceSum(qk_tmp);

  if (threadIdx.x == 0) {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  if (threadIdx.x < seq_len)
    qk_buf_[threadIdx.x + qk_offset] = (qk_tmp / s_sum);
}

template <>
void GPUSoftmaxMask(float* qk_buf, const float* attr_mask, int64_t batch_size,
                    int64_t head_num, int64_t seq_len, float scale,
                    hipStream_t stream) {
  dim3 block, grid;
  int blk_size;
  int high_dim_size = batch_size * head_num * seq_len;

  // block size must be 32x, so warp reduce can work
  block.x = (seq_len + 31) / 32 * 32;
  blk_size = 4;
  // In the senario of BERT inference, high_dim_size is 4x because head_num is
  // 12
  if (high_dim_size < 40 * 12 || high_dim_size % blk_size != 0) {
    grid.x = high_dim_size;
    softmax_kernel_noblk<<<grid, block, 0, stream>>>(
        qk_buf, attr_mask, batch_size, head_num, seq_len, scale);
  } else {
    grid.x = high_dim_size / blk_size;
    softmax_kernel_unroll4<<<grid, block, 0, stream>>>(
        qk_buf, attr_mask, batch_size, head_num, seq_len, scale, blk_size);
  }
}

}  // namespace kernels
}  // namespace layers
}  // namespace fast_transformers
