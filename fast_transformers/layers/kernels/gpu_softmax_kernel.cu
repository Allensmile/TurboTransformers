#include "hip/hip_runtime.h"
#include "fast_transformers/layers/kernels/gpu_softmax_kernel.h"
#include <immintrin.h>
#include <numeric>
#include <hip/hip_runtime.h>

//copy from https://github.com/NVIDIA/DeepLearningExamples/tree/master/FasterTransformer

namespace fast_transformers {
namespace layers {
namespace kernels {

/**
 * Multi-head attetion open sourced
 */
#define FINAL_MASK 0xffffffff

template <typename T>
__inline__ __device__
T warpReduceSum(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1)
    val += __shfl_xor_sync(FINAL_MASK, val, mask, 32);
  return val;
}

/* Calculate the sum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceSum(T val)
{
  static __shared__ T shared[32];
  int lane = threadIdx.x & 0x1f;
  int wid = threadIdx.x >> 5;

  val = warpReduceSum<T>(val);

  if(lane == 0)
    shared[wid] = val;

  __syncthreads();

  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : (T)(0.0f);
  val = warpReduceSum<T>(val);

  return val;
}

template <typename T>
  __inline__ __device__
T warpReduceMax(T val)
{
  for(int mask = 16; mask > 0; mask >>= 1)
    val = max(val, __shfl_xor_sync(FINAL_MASK, val, mask, 32));
  return val;
}

/* Calculate the maximum of all elements in a block */
template <typename T>
  __inline__ __device__
T blockReduceMax(T val)
{
  static __shared__ T shared[32];
  int lane = threadIdx.x & 0x1f; // in-warp idx
  int wid = threadIdx.x >> 5;  // warp idx

  val = warpReduceMax(val); // get maxx in each warp

  if(lane == 0) // record in-warp maxx by warp Idx
    shared[wid] = val;

  __syncthreads();


  val = (threadIdx.x < (blockDim.x >> 5 )) ? shared[lane] : -1e20f;
  val = warpReduceMax(val);

  return val;
}


template <typename T>
__global__
void softmax_kernel(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num, const int seq_len,
  const T scaler)
{
    int batch_id = blockIdx.x / head_num;
    int qk_offset = blockIdx.x * seq_len * seq_len;

    __shared__ float s_sum, s_max;

    for(int i = 0; i < seq_len; ++i)
    {
      float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
      float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x % seq_len + batch_id * seq_len
      ] : 0.0f;

      //mask_val = (1.0f - mask_val) * -10000.0f;

      float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scaler + mask_val): -1e20f;

      float max_val = blockReduceMax<float>(tmp);

      if(threadIdx.x == 0)
        s_max = max_val;
      __syncthreads();

      qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

      float sum_val = blockReduceSum<float>(qk);

      if(threadIdx.x == 0)
      {
        s_sum = sum_val + 1e-6f;
      }
      __syncthreads();

      if(threadIdx.x < seq_len)
        qk_buf_[threadIdx.x + qk_offset] = (T)(qk / s_sum);

      qk_offset += seq_len;
    }
}


template <typename T>
__global__
void softmax_kernel_v2(T* qk_buf_, const T* attr_mask, const int batch_size, const int head_num,
  const int seq_len, const float scaler)
{
    int batch_id = blockIdx.x / head_num / seq_len;
    int seq_id = blockIdx.x % seq_len;
    int qk_offset = blockIdx.x * seq_len;
    //int mask_offset = batch_id * seq_len * seq_len + seq_id * seq_len;
    int mask_offset = batch_id * seq_len;

    __shared__ float s_sum, s_max;

    float qk = threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
    //float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] :0.0f;
    float mask_val = threadIdx.x < seq_len ? (float)attr_mask[threadIdx.x + mask_offset] :0.0f;

    //mask_val = (1.0f - mask_val) * -10000.0f;

    float tmp = threadIdx.x < seq_len ? (float)(qk * (float)scaler + mask_val) : -1e20f;
    float max_val = blockReduceMax<float>(tmp);
    if(threadIdx.x == 0)
      s_max = max_val;
    __syncthreads();

    float qk_tmp = threadIdx.x < seq_len ? __expf((float)(tmp - s_max)) : 0.0f;
    float sum_val = blockReduceSum<float>(qk_tmp);

    if(threadIdx.x == 0)
    {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if(threadIdx.x < seq_len)
      qk_buf_[threadIdx.x + qk_offset] = (T)(qk_tmp / s_sum);
}

void GPUSoftmaxMask(float* qk_buf, const float* attr_mask,
                        int64_t batch_size, int64_t head_num, int64_t seq_len,
                        float scale, hipStream_t stream) {
  dim3 block, grid;
  if(seq_len <= 32)
    block.x = 32;
  else if(seq_len > 32 && seq_len <= 64)
    block.x = 64;
  else if(seq_len > 64 && seq_len <= 128)
    block.x = 128;
  else if(seq_len > 128 && seq_len <= 256)
    block.x = 256;
  else if(seq_len > 256 && seq_len <= 512)
    block.x = 512;
  else
    block.x = 1024;
  //assert(n > 1024);
  if(batch_size * head_num <= 120)
  {
    grid.x = batch_size * head_num * seq_len;
    softmax_kernel_v2<float><<<grid, block, 0, stream>>>(qk_buf, attr_mask, batch_size, head_num, seq_len, scale);
  }
  else
  {
    grid.x = batch_size * head_num;
    softmax_kernel<float><<<grid, block, 0, stream>>>(qk_buf, attr_mask, batch_size, head_num, seq_len, scale);
  }
}

#undef FINAL_MASK

}  // namespace kernels
}  // namespace layers
}  // namespace fast_transformers
