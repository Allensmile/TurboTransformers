#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <numeric>
#include "fast_transformers/layers/kernels/gpu_common.h"
#include "fast_transformers/layers/kernels/gpu_softmax_kernel.h"

namespace fast_transformers {
namespace layers {
namespace kernels {

// blk_size == 4
__global__ void softmax_kernel_blk4(float* qk_buf_, const float* attr_mask,
                                    int batch_size, int head_num, int seq_len,
                                    float scaler, int blk_size) {
  int batch_id = blockIdx.x * blk_size / seq_len / head_num;
  int tid = threadIdx.x;
  int qk_offset = tid + blockIdx.x * seq_len * blk_size;

#define MAX_BLK_INNNER_SIZE 4
  static __shared__ float s_sum[MAX_BLK_INNNER_SIZE];
  float qk_list[MAX_BLK_INNNER_SIZE];
  float qk_sum_list[MAX_BLK_INNNER_SIZE];
  const int max_blk_inner_size = MAX_BLK_INNNER_SIZE;

  float mask_val = tid < seq_len
                       ? (float)attr_mask[tid % seq_len + batch_id * seq_len]
                       : 0.0f;
  float qk;
  int i;
  for (i = 0; i < blk_size / max_blk_inner_size * max_blk_inner_size;
       i += max_blk_inner_size) {
    int blk_size_inner = max_blk_inner_size;
    if (threadIdx.x < seq_len) {
      int qk_buf_offset = qk_offset + i * seq_len;
      qk = (float)qk_buf_[qk_buf_offset];
      qk_sum_list[0] = qk_list[0] =
          __expf((float)(qk * (float)scaler + mask_val));

      qk = (float)qk_buf_[qk_buf_offset + seq_len];
      qk_sum_list[1] = qk_list[1] =
          __expf((float)(qk * (float)scaler + mask_val));

      qk = (float)qk_buf_[qk_buf_offset + 2 * seq_len];
      qk_sum_list[2] = qk_list[2] =
          __expf((float)(qk * (float)scaler + mask_val));

      qk = (float)qk_buf_[qk_buf_offset + 3 * seq_len];
      qk_sum_list[3] = qk_list[3] =
          __expf((float)(qk * (float)scaler + mask_val));
    } else {
      qk_sum_list[0] = qk_list[0] = 0.0;
      qk_sum_list[1] = qk_list[1] = 0.0;
      qk_sum_list[2] = qk_list[2] = 0.0;
      qk_sum_list[3] = qk_list[3] = 0.0;
    }
    blockReduceSum_Elem4(qk_sum_list);
    if (tid == 0) {
      s_sum[0] = qk_sum_list[0] + 1e-6f;
      s_sum[1] = qk_sum_list[1] + 1e-6f;
      s_sum[2] = qk_sum_list[2] + 1e-6f;
      s_sum[3] = qk_sum_list[3] + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len) {
      qk_buf_[qk_offset + seq_len * (i + 0)] = (float)(qk_list[0] / s_sum[0]);
      qk_buf_[qk_offset + seq_len * (i + 1)] = (float)(qk_list[1] / s_sum[1]);
      qk_buf_[qk_offset + seq_len * (i + 2)] = (float)(qk_list[2] / s_sum[2]);
      qk_buf_[qk_offset + seq_len * (i + 3)] = (float)(qk_list[3] / s_sum[3]);
    }  // endif
  }    // for i

  // dealing with the reminding lines
  int blk_size_inner = seq_len % max_blk_inner_size;
  if (blk_size_inner == 0) return;
  if (threadIdx.x < seq_len) {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk = (float)qk_buf_[qk_offset + (i + j) * seq_len];
      qk_sum_list[j] = qk_list[j] =
          __expf((float)(qk * (float)scaler + mask_val));
    }
  } else {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk_sum_list[j] = qk_list[j] = 0.0;
    }
  }
  for (int j = 0; j < blk_size_inner; ++j) {
    qk_sum_list[j] = blockReduceSum(qk_list[j]);
    __syncthreads();
  }
  if (tid == 0) {
    for (int j = 0; j < blk_size_inner; ++j) {
      s_sum[j] = qk_sum_list[j] + 1e-6f;
    }
  }
  __syncthreads();
  if (threadIdx.x < seq_len) {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk_buf_[qk_offset + seq_len * (i + j)] = (float)(qk_list[j] / s_sum[j]);
    }
  }  // endif
#undef MAX_BLK_INNNER_SIZE
}

__global__ void softmax_kernel_blk5(float* qk_buf_, const float* attr_mask,
                                    int batch_size, int head_num, int seq_len,
                                    float scaler, int blk_size) {
  int batch_id = blockIdx.x * blk_size / seq_len / head_num;
  int tid = threadIdx.x;
  int qk_offset = tid + blockIdx.x * seq_len * blk_size;

#define MAX_BLK_INNNER_SIZE 5
  static __shared__ float s_sum[MAX_BLK_INNNER_SIZE];
  float qk_list[MAX_BLK_INNNER_SIZE];
  float qk_sum_list[MAX_BLK_INNNER_SIZE];
  const int max_blk_inner_size = MAX_BLK_INNNER_SIZE;

  float mask_val = tid < seq_len
                       ? (float)attr_mask[tid % seq_len + batch_id * seq_len]
                       : 0.0f;
  float qk;
  int i;
  for (i = 0; i < blk_size / max_blk_inner_size * max_blk_inner_size;
       i += max_blk_inner_size) {
    int blk_size_inner = max_blk_inner_size;
    if (threadIdx.x < seq_len) {
      int qk_buf_offset = qk_offset + i * seq_len;
      qk = (float)qk_buf_[qk_buf_offset];
      qk_sum_list[0] = qk_list[0] =
          __expf((float)(qk * (float)scaler + mask_val));

      qk = (float)qk_buf_[qk_buf_offset + seq_len];
      qk_sum_list[1] = qk_list[1] =
          __expf((float)(qk * (float)scaler + mask_val));

      qk = (float)qk_buf_[qk_buf_offset + 2 * seq_len];
      qk_sum_list[2] = qk_list[2] =
          __expf((float)(qk * (float)scaler + mask_val));

      qk = (float)qk_buf_[qk_buf_offset + 3 * seq_len];
      qk_sum_list[3] = qk_list[3] =
          __expf((float)(qk * (float)scaler + mask_val));

      qk = (float)qk_buf_[qk_buf_offset + 4 * seq_len];
      qk_sum_list[4] = qk_list[4] =
          __expf((float)(qk * (float)scaler + mask_val));
    } else {
      qk_sum_list[0] = qk_list[0] = 0.0;
      qk_sum_list[1] = qk_list[1] = 0.0;
      qk_sum_list[2] = qk_list[2] = 0.0;
      qk_sum_list[3] = qk_list[3] = 0.0;
      qk_sum_list[4] = qk_list[4] = 0.0;
    }
    blockReduceSum_Elem5(qk_sum_list, blk_size_inner);
    if (tid == 0) {
      s_sum[0] = qk_sum_list[0] + 1e-6f;
      s_sum[1] = qk_sum_list[1] + 1e-6f;
      s_sum[2] = qk_sum_list[2] + 1e-6f;
      s_sum[3] = qk_sum_list[3] + 1e-6f;
      s_sum[4] = qk_sum_list[4] + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len) {
      qk_buf_[qk_offset + seq_len * (i + 0)] = (float)(qk_list[0] / s_sum[0]);
      qk_buf_[qk_offset + seq_len * (i + 1)] = (float)(qk_list[1] / s_sum[1]);
      qk_buf_[qk_offset + seq_len * (i + 2)] = (float)(qk_list[2] / s_sum[2]);
      qk_buf_[qk_offset + seq_len * (i + 3)] = (float)(qk_list[3] / s_sum[3]);
      qk_buf_[qk_offset + seq_len * (i + 4)] = (float)(qk_list[4] / s_sum[4]);
    }  // endif
  }    // for i

  // dealing with reminding lines
  int blk_size_inner = seq_len % max_blk_inner_size;
  if (blk_size_inner == 0) return;
  if (threadIdx.x < seq_len) {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk = (float)qk_buf_[qk_offset + (i + j) * seq_len];
      qk_sum_list[j] = qk_list[j] =
          __expf((float)(qk * (float)scaler + mask_val));
    }
  } else {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk_sum_list[j] = qk_list[j] = 0.0;
    }
  }
  for (int j = 0; j < blk_size_inner; ++j) {
    qk_sum_list[j] = blockReduceSum(qk_list[j]);
    __syncthreads();
  }
  if (tid == 0) {
    for (int j = 0; j < blk_size_inner; ++j) {
      s_sum[j] = qk_sum_list[j] + 1e-6f;
    }
  }
  __syncthreads();
  if (threadIdx.x < seq_len) {
    for (int j = 0; j < blk_size_inner; ++j) {
      qk_buf_[qk_offset + seq_len * (i + j)] = (float)(qk_list[j] / s_sum[j]);
    }
  }  // endif
#undef MAX_BLK_INNNER_SIZE
}

// the original softmax implementation from nvidia but with no max-trick.
// baseline for comparison
__global__ void softmax_kernel_nomax_baseline(float* qk_buf_,
                                              const float* attr_mask,
                                              int batch_size, int head_num,
                                              int seq_len, float scaler,
                                              int blk_size) {
  int batch_id = blockIdx.x * blk_size / seq_len / head_num;
  int qk_offset = blockIdx.x * seq_len * seq_len;

  __shared__ float s_sum;

  for (int i = 0; i < blk_size; ++i) {
    float qk =
        threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
    float mask_val =
        threadIdx.x < seq_len
            ? (float)attr_mask[threadIdx.x % seq_len + batch_id * seq_len]
            : 0.0f;

    // mask_val = (1.0f - mask_val) * -10000.0f;
    qk = threadIdx.x < seq_len ? __expf((float)(qk * (float)scaler + mask_val))
                               : 0.0f;

    float sum_val = blockReduceSum(qk);

    if (threadIdx.x == 0) {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf_[threadIdx.x + qk_offset] = (float)(qk / s_sum);

    qk_offset += seq_len;
  }
}

// jiaruifang optimized version for softmax no max-trick
__global__ void softmax_kernel_nomax(float* qk_buf_, const float* attr_mask,
                                     int batch_size, int head_num, int seq_len,
                                     float scaler, int blk_size) {
  int batch_id = blockIdx.x * blk_size / seq_len / head_num;
  int qk_offset = blockIdx.x * seq_len * blk_size;

  __shared__ float s_sum;

  float mask_val =
      threadIdx.x < seq_len
          ? (float)attr_mask[threadIdx.x % seq_len + batch_id * seq_len]
          : 0.0f;

  float qk;
  for (int i = 0; i < blk_size; ++i) {
    if (threadIdx.x < seq_len) {
      qk = (float)qk_buf_[threadIdx.x + qk_offset];
      qk = __expf((float)(qk * (float)scaler + mask_val));
    } else {
      qk = 0.0;
    }

    float sum_val = blockReduceSum(qk);

    if (threadIdx.x == 0) {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf_[threadIdx.x + qk_offset] = (float)(qk / s_sum);

    qk_offset += seq_len;
  }
}

// nvidia version seq_len as block size on the high dimension of qk_buf_
__global__ void softmax_kernel(float* qk_buf_, const float* attr_mask,
                               int batch_size, int head_num, int seq_len,
                               float scaler) {
  int batch_id = blockIdx.x / head_num;
  int qk_offset = blockIdx.x * seq_len * seq_len;

  __shared__ float s_sum, s_max;
  float mask_val =
      threadIdx.x < seq_len
          ? (float)attr_mask[threadIdx.x % seq_len + batch_id * seq_len]
          : 0.0f;

  for (int i = 0; i < seq_len; ++i) {
    float qk =
        threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
    // mask_val = (1.0f - mask_val) * -10000.0f;
    float tmp =
        threadIdx.x < seq_len ? (float)(qk * (float)scaler + mask_val) : -1e20f;

    float max_val = blockReduceMax(tmp);

    if (threadIdx.x == 0) s_max = max_val;
    __syncthreads();

    qk = threadIdx.x < seq_len ? __expf(tmp - s_max) : 0.0f;

    float sum_val = blockReduceSum(qk);

    if (threadIdx.x == 0) {
      s_sum = sum_val + 1e-6f;
    }
    __syncthreads();

    if (threadIdx.x < seq_len)
      qk_buf_[threadIdx.x + qk_offset] = (float)(qk / s_sum);

    qk_offset += seq_len;
  }
}

// nvidia version block size on the high dimension is always 1, may lead to
// low occupancy
__global__ void softmax_kernel_blk1(float* qk_buf_, const float* attr_mask,
                                    const int batch_size, const int head_num,
                                    const int seq_len, const float scaler) {
  int batch_id = blockIdx.x / head_num / seq_len;
  int qk_offset = blockIdx.x * seq_len;
  int mask_offset = batch_id * seq_len;

  __shared__ float s_sum, s_max;

  float qk =
      threadIdx.x < seq_len ? (float)qk_buf_[threadIdx.x + qk_offset] : 0.0f;
  float mask_val = threadIdx.x < seq_len
                       ? (float)attr_mask[threadIdx.x + mask_offset]
                       : 0.0f;

  // mask_val = (1.0f - mask_val) * -10000.0f;
  float tmp =
      threadIdx.x < seq_len ? (float)(qk * (float)scaler + mask_val) : -1e20f;
  float max_val = blockReduceMax(tmp);
  if (threadIdx.x == 0) s_max = max_val;
  __syncthreads();

  float qk_tmp = threadIdx.x < seq_len ? __expf((float)(tmp - s_max)) : 0.0f;
  float sum_val = blockReduceSum(qk_tmp);

  if (threadIdx.x == 0) {
    s_sum = sum_val + 1e-6f;
  }
  __syncthreads();

  if (threadIdx.x < seq_len)
    qk_buf_[threadIdx.x + qk_offset] = (float)(qk_tmp / s_sum);
}

template <>
void GPUSoftmaxMask(float* qk_buf, const float* attr_mask, int64_t batch_size,
                    int64_t head_num, int64_t seq_len, float scale,
                    hipStream_t stream) {
  dim3 block, grid;
  int blk_size;
  int high_dim_size = batch_size * head_num * seq_len;

  // block size must be 32x, so warp reduce can work
  block.x = (seq_len + 31) / 32 * 32;
  if (batch_size * head_num <= 120) {
    blk_size = 5;
    // In the senario of BERT inference, high_dim_size is 4x because head_num is
    // 12
    if (high_dim_size % blk_size == 0) {
      grid.x = high_dim_size / blk_size;
      if (blk_size == 5) {
        softmax_kernel_blk5<<<grid, block, 0, stream>>>(
            qk_buf, attr_mask, batch_size, head_num, seq_len, scale, blk_size);

      } else if (blk_size == 4) {
        softmax_kernel_blk4<<<grid, block, 0, stream>>>(
            qk_buf, attr_mask, batch_size, head_num, seq_len, scale, blk_size);
      } else {
        printf("blk_size %d is not supported\n", blk_size);
      }
    } else {
      grid.x = high_dim_size;
      softmax_kernel_blk1<<<grid, block, 0, stream>>>(
          qk_buf, attr_mask, batch_size, head_num, seq_len, scale);
    }
  } else {
    blk_size = seq_len;
    grid.x = batch_size * head_num * seq_len / blk_size;
    softmax_kernel_blk5<<<grid, block, 0, stream>>>(
        qk_buf, attr_mask, batch_size, head_num, seq_len, scale, blk_size);
  }
}

}  // namespace kernels
}  // namespace layers
}  // namespace fast_transformers
