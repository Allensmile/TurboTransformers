#include "hip/hip_runtime.h"
// Copyright 2020 Tencent
// 
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
// 
//     http://www.apache.org/licenses/LICENSE-2.0
// 
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <numeric>

#include "fast_transformers/layers/kernels/gpu_activation_kernel.h"
#include "ide_macro.h"

namespace fast_transformers {
namespace layers {
namespace kernels {

static __inline__ __device__ float gelu(float x) {
  float cdf =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
  return x * cdf;
}

static __global__ void add_bias_act(float* out, const float* bias,
                                    int batch_size, int feature_dim) {
  float val, reg_bias;

  int row_id = blockIdx.x;
  int ite = feature_dim / blockDim.x;
  int tid = threadIdx.x;

  for (int i = 0; i < ite; ++i) {
    reg_bias = __ldg(&bias[i * blockDim.x + tid]);
    row_id = blockIdx.x;

    while (row_id < batch_size) {
      val = out[tid + i * blockDim.x + row_id * feature_dim] + reg_bias;
      out[tid + i * blockDim.x + row_id * feature_dim] = gelu(val);
      row_id += gridDim.x;
    }
  }
}
template <>
void GPUAddBiasGeLUActKernel(const float* bias_data, float* out_data,
                             int64_t batch_size, int64_t feature_dim,
                             hipStream_t stream) {
  dim3 grid(batch_size / 4);
  dim3 block(feature_dim / 4);
  if (feature_dim / 4 > 1024) {
    throw std::runtime_error(
        "GPUAddBiasGeLUActKernel thread block size large than 1024");
  }
  add_bias_act<<<grid, block, 0, stream>>>(out_data, bias_data, batch_size,
                                           feature_dim);
}

}  // namespace kernels
}  // namespace layers
}  // namespace fast_transformers
