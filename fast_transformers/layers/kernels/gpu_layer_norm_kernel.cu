#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <immintrin.h>
#include <numeric>
#include "fast_transformers/layers/kernels/gpu_common.h"
#include "fast_transformers/layers/kernels/gpu_layer_norm_kernel.h"

namespace fast_transformers {
namespace layers {
namespace kernels {

inline __device__ void get_mean_variance(float val, float* s_mean,
                                         float* s_variance, int n, int tid) {
  float sum1 = val, sum2 = val * val;
  blockReduceSumTwoElemInline(&sum1, &sum2);
  float mean = sum1 / n;
  float mean_2 = sum2 / n;

  if (tid == 0) {
    *s_mean = mean;
    *s_variance = rsqrtf(mean_2 - mean * mean + 1e-6f);
  }
  __syncthreads();
}

static __global__ void add_bias_input_layernorm(float* out, const float* input,
                                                const float* bias,
                                                const float* gamma,
                                                const float* beta, int m,
                                                int n) {
  __shared__ float s_mean;
  __shared__ float s_variance;
  float mean = 0.0f;
  float variance = 0.0f;

  float local_out = 0.0f;
  for (int i = tid; i < n; i += blockDim.x)
    local_out +=
        out[blockIdx.x * n + i] + input[blockIdx.x * n + i] + __ldg(&bias[i]);

  get_mean_variance(local_out, &s_mean, &s_variance, n, threadIdx.x);
  for (int i = tid; i < n; i += blockDim.x) {
    out[blockIdx.x * n + i] =
        (local_out - s_mean) * s_variance * __ldg(&gamma[i]) + __ldg(&beta[i]);
  }
}

template <>
void GPUAddBiasLayerNorm(float* out, const float* input, const float* bias,
                         const float* gamma, const float* beta, int m, int n,
                         hipStream_t stream) {
  dim3 grid(m);
  dim3 block(n);
  if (n > 1024) {
    throw std::runtime_error(
        "GPUAddBiasLayerNorm thread block size large than 1024");
  }
  add_bias_input_layernorm<<<grid, block, 0, stream>>>(out, input, bias, gamma,
                                                       beta, m, n);
}

static __global__ void layernorm(float* out, const float* gamma,
                                 const float* beta, int m, int n) {
  __shared__ float s_mean;
  __shared__ float s_variance;

  float local_out = 0.0f;
  for (int i = tid; i < n; i += blockDim.x)
    local_out += (out[blockIdx.x * n + i]);

  get_mean_variance(local_out, &s_mean, &s_variance, n, threadIdx.x);

  for (int i = tid; i < n; i += blockDim.x) {
    out[blockIdx.x * n + i] =
        (local_out - s_mean) * s_variance * __ldg(&gamma[i]) + __ldg(&beta[i]);
  }
}

template <>
void GPULayerNorm(float* out, const float* gamma, const float* beta, int m,
                  int n, hipStream_t stream) {
  dim3 grid(m);
  dim3 block(n);
  if (n > 1024) {
    throw std::runtime_error(
        "GPUAddBiasLayerNorm thread block size large than 1024");
  }
  layernorm<<<grid, block, 0, stream>>>(out, gamma, beta, m, n);
}

}  // namespace kernels
}  // namespace layers
}  // namespace fast_transformers
