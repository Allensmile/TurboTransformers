#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <immintrin.h>

#include <hipcub/hipcub.hpp>
#include <numeric>

#include "fast_transformers/layers/kernels/gpu_block_reduce.h"
#include "fast_transformers/layers/kernels/gpu_layer_norm_kernel.h"

namespace fast_transformers {
namespace layers {
namespace kernels {

namespace {
template <typename T>
struct DataPair {
  __device__ __forceinline__ DataPair() {}
  __device__ __forceinline__ DataPair(const T& first, const T& second)
      : first(first), second(second) {}

  T first;
  T second;
};

template <typename T>
struct DataPairAddFunc {
  __device__ __forceinline__ DataPair<T> operator()(const DataPair<T>& p1,
                                                    const DataPair<T>& p2) {
    return DataPair<T>(p1.first + p2.first, p1.second + p2.second);
  }
};

template <bool isAdd, int BlockDim, typename T>
__global__ void cub_layer_norm_kernel(T* out, const T* input, const T* bias,
                                      const T* gamma, const T* beta, int m,
                                      int n) {
  using CubBlockReduce = hipcub::BlockReduce<DataPair<float>, BlockDim>;
  __shared__ typename CubBlockReduce::TempStorage temp_storage;
  __shared__ T s_mean;
  __shared__ T s_variance;

  int tid = threadIdx.x;
  T val1 = 0.0f, val2 = 0.0f;
  if (tid < n) {
    T tmp = input[blockIdx.x * n + tid];
    if (isAdd) {
      tmp += out[blockIdx.x * n + tid] + __ldg(&bias[tid]);
    }
    val1 = tmp;
    val2 = tmp * tmp;
  }

  auto pair =
      CubBlockReduce(temp_storage)
          .Reduce(DataPair<float>(val1, val2), DataPairAddFunc<float>());

  if (tid == 0) {
    s_mean = pair.first / n;
    s_variance = rsqrtf(pair.second / n - s_mean * s_mean + 1e-6f);
  }
  __syncthreads();

  if (tid < n) {
    out[blockIdx.x * n + tid] =
        (val1 - s_mean) * s_variance * __ldg(&gamma[tid]) + __ldg(&beta[tid]);
  }
}
}  // namespace

template <bool AddBias>
static __global__ void layer_norm_kernel(float* out, const float* input,
                                         const float* bias, const float* gamma,
                                         const float* beta, int m, int n) {
  int tid = threadIdx.x;
  int offset = blockIdx.x * n + tid;
  __shared__ float s_mean;
  __shared__ float s_variance;

  float local_out = 0.0f;
  if (AddBias) {
    local_out = out[offset] + input[offset] + __ldg(&bias[tid]);
  } else {
    local_out = out[offset];
  }

  float sum1 = local_out, sum2 = local_out * local_out;
  blockReduceSum_Elem2(&sum1, &sum2);

  if (tid == 0) {
    float mean = sum1 / n;
    float mean_2 = sum2 / n;
    s_mean = mean;
    s_variance = rsqrtf(mean_2 - mean * mean + 1e-6f);
  }
  __syncthreads();

  out[offset] = (local_out - s_mean) * s_variance * __ldg(&gamma[tid]) +
                __ldg(&beta[tid]);
}

/*
template <bool AddBias, typename T>
void GPULayerNorm(T *out, const T *input, const T *bias, const T *gamma,
                  const T *beta, int m, int n, hipStream_t stream) {
  if (n > 1024) {
    throw std::runtime_error("GPULayerNorm thread block size large than 1024.");
  }
  dim3 grid(m);
  dim3 block(1 << static_cast<int>(std::ceil(std::log2f(n))));

#define LayerNormKernelCase(AddMode, BlockDim, ...) \
  case (BlockDim):                                  \
    cub_layer_norm_kernel<(AddMode), (BlockDim)>    \
        <<<grid, block, 0, stream>>>(__VA_ARGS__);  \
    break

  switch (block.x) {
    LayerNormKernelCase(AddBias, 1024, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 512, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 128, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 64, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 32, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 16, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 8, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 4, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 2, out, input, bias, gamma, beta, m, n);
    LayerNormKernelCase(AddBias, 1, out, input, bias, gamma, beta, m, n);
  }
#undef LayerNormKernelCase
}
*/

template <bool AddBias, typename T>
void GPULayerNorm(T* out, const T* input, const T* bias, const T* gamma,
                  const T* beta, int m, int n, hipStream_t stream) {
  dim3 block(n);
  if (block.x > 1024) {
    throw std::runtime_error("GPULayerNorm thread block size large than 1024");
  }
  dim3 grid(m);
  layer_norm_kernel<AddBias>
      <<<grid, block, 0, stream>>>(out, input, bias, gamma, beta, m, n);
}

template void GPULayerNorm<true>(float* out, const float* input,
                                 const float* bias, const float* gamma,
                                 const float* beta, int m, int n,
                                 hipStream_t stream);
template void GPULayerNorm<false>(float* out, const float* input,
                                  const float* bias, const float* gamma,
                                  const float* beta, int m, int n,
                                  hipStream_t stream);
}  // namespace kernels
}  // namespace layers
}  // namespace fast_transformers
