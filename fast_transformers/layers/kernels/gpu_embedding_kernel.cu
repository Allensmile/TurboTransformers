#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <numeric>
#include "fast_transformers/layers/kernels/gpu_embedding_kernel.h"

namespace fast_transformers {
namespace layers {
namespace kernels {

template <bool IsAdd>
static __global__ void lookup(float* dst, const float* embedding_table,
                              const int64_t* ids, int64_t vocab_size) {
  int64_t id = ids[blockIdx.x];
  int hidden_idx = threadIdx.x;
  int hidden_size = blockDim.x;
  // TODO(jiaruifang): There should have a checker to check the range of id.
  if (id >= vocab_size) {
    asm("trap;");
  }

  float val = __ldg(&embedding_table[id * hidden_size + hidden_idx]);
  if (IsAdd) {
    dst[blockIdx.x * hidden_size + hidden_idx] += val;
  } else {
    dst[blockIdx.x * hidden_size + hidden_idx] = val;
  }
}

void GPULookupKernel(float* dst, const float* embedding_table,
                     const int64_t* ids, int64_t vocab_size,
                     int64_t hidden_size, int64_t num_ids, bool is_add,
                     hipStream_t stream) {
  dim3 grid(num_ids);
  dim3 block(hidden_size);
  if (block.x > 1024) {
    throw std::runtime_error(
        "GPULookupKernel currently does not support a hidden_size larger than "
        "1024");
  }
  if (is_add) {
    lookup<true>
        <<<grid, block, 0, stream>>>(dst, embedding_table, ids, vocab_size);
  } else {
    lookup<false>
        <<<grid, block, 0, stream>>>(dst, embedding_table, ids, vocab_size);
  }
}

}  // namespace kernels
}  // namespace layers
}  // namespace fast_transformers
