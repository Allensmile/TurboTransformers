#include "hip/hip_runtime.h"
// Copyright 2020 Tencent
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <numeric>

#include "ide_macro.h"
#include "turbo_transformers/core/half.h"
#include "turbo_transformers/layers/kernels/gpu_activation_kernel.h"

namespace turbo_transformers {
namespace layers {
namespace kernels {

template <typename T>
static __inline__ __device__ T add(const T& a, const T& b) {
  return a + b;
}

static __inline__ __device__ __half add(const __half& a, const __half& b) {
  return __hadd(a, b);
}

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 600
static __inline__ __device__ __half2 add(const __half2& a, const __half2& b) {
  return __hadd2(a, b);
}
#endif

template <typename T, ActivationType aT>
__device__ T ActvationOp(const T& x);

template <>
__device__ float ActvationOp<float, ActivationType::Gelu>(const float& x) {
  float cdf =
      0.5f *
      (1.0f + tanhf((0.7978845608028654f * (x + 0.044715f * x * x * x))));
  return x * cdf;
}

template <>
__device__ __half ActvationOp<__half, ActivationType::Gelu>(const __half& x) {
  float x_f = __half2float(x);
  return __float2half(ActvationOp<float, ActivationType::Gelu>(x_f));
}

template <>
__device__ float ActvationOp<float, ActivationType::Tanh>(const float& x) {
  return tanhf(x);
}

template <>
__device__ __half ActvationOp<__half, ActivationType::Tanh>(const __half& x) {
  float x_f = __half2float(x);
  return __float2half(ActvationOp<float, ActivationType::Tanh>(x_f));
}

template <typename T, ActivationType aT>
static __global__ void add_bias_act(T* out, const T* bias, int batch_size,
                                    int feature_dim) {
  T val, reg_bias;

  int row_id;
  int elem_per_thread = (feature_dim + blockDim.x - 1) / blockDim.x;
  int tid = threadIdx.x;

  for (int i = 0; i < elem_per_thread; ++i) {
    int offset = i * blockDim.x + tid;
    if (offset < feature_dim) {
      reg_bias = __ldg(&bias[offset]);
      row_id = blockIdx.x;
      val = add(out[offset + row_id * feature_dim], reg_bias);
      out[offset + row_id * feature_dim] = ActvationOp<T, aT>(val);
    }
  }
}

template <typename T, ActivationType aT>
void GPUAddBiasActKernel(const T* bias_data, T* out_data, int64_t batch_size,
                         int64_t feature_dim, hipStream_t stream) {
  dim3 grid(batch_size);
  int block_size = min(1024, (int)(feature_dim / 4));
  dim3 block(block_size);
  add_bias_act<T, aT><<<grid, block, 0, stream>>>(out_data, bias_data,
                                                  batch_size, feature_dim);
}

template void GPUAddBiasActKernel<float, ActivationType::Gelu>(
    const float* bias_data, float* out_data, int64_t batch_size,
    int64_t feature_dim, hipStream_t stream);

template void GPUAddBiasActKernel<float, ActivationType::Tanh>(
    const float* bias_data, float* out_data, int64_t batch_size,
    int64_t feature_dim, hipStream_t stream);

template void GPUAddBiasActKernel<half, ActivationType::Gelu>(
    const half* bias_data, half* out_data, int64_t batch_size,
    int64_t feature_dim, hipStream_t stream);

}  // namespace kernels
}  // namespace layers
}  // namespace turbo_transformers
